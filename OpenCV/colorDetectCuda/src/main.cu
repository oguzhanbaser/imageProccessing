#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdio.h>
#include <string>

#define USE_GPU                         //if you do dnot want to use GPU make comment this line

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/video/video.hpp>
#ifdef USE_GPU
#include <opencv2/gpu/gpu.hpp>
#include <opencv2/gpu/devmem2d.hpp>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#endif

using namespace std;
using namespace cv;

#define IMG_W 640
#define IMG_H 480

#define MAX_NUM_OBJECTS 10

#ifdef USE_GPU
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
            file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
            file, line, hipGetErrorString(err));
        exit(-1);
    }


    err = hipDeviceSynchronize();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
            file, line, hipGetErrorString(err));
        exit(-1);
    }
#endif

    return;
}

extern "C"__global__ void gpu_inRange(const gpu::PtrStepSz<uchar3> p_src, gpu::PtrStepSzb p_dst,
                                      int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < p_src.cols && y < p_src.rows)
    {
        uchar3 v = p_src(y, x);
        if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
        {
            p_dst(y, x) = 255;
        }else{
            p_dst(y, x) = 0;
        }
    }
}

#endif

void trackObject(Mat &threshold, Mat &orjImage, Scalar __s)
{
    vector< vector<Point> > contours;
    vector<Vec4i> hierarchy;
    Mat temp;

    findContours(threshold, contours, hierarchy, CV_RETR_EXTERNAL, CV_CHAIN_APPROX_SIMPLE);		//dış kenarları bul

    if (hierarchy.size() > 0)
    {
        int numObjects = hierarchy.size();

        if (numObjects < MAX_NUM_OBJECTS)
        {
            int biggestArea = 0, biggestIndex = -1;
            for (int i = 0; i >= 0; i = hierarchy[i][0])
            {
                Moments moment = moments((cv::Mat)contours[i]);
                double area = moment.m00;
                //feedback
                //cout << area << endl;
                if(area > biggestArea)
                {
                    biggestIndex = i;
                    biggestArea = area;
                }

                //circle(orjImage, Point(xPos, yPos), );
                for (int i = 0; i < contours.size(); i++)
                    drawContours(orjImage, contours, i, __s, 3, 8, hierarchy);

            }

            if(biggestIndex >= 0)
            {
                Moments moment = moments((cv::Mat)contours[biggestIndex]);
                double area = moment.m00;

                int xPos = moment.m10 / area;
                int yPos = moment.m01 / area;
                //cout << moment.m10 << "\t" << moment.m01 << endl;

                cout << "Biggest obj: X:" << xPos << "\tY: " << yPos << endl;
            }
        }
        else{
            putText(orjImage, "Cok Fazla Obje Var!", Point(10, 50), 1, 2, Scalar(0, 0, 255));
        }
    }

    //cout << hierarchy.data() << "\t" << hierarchy.size() << endl;
}


int main(int argc, char *argv[])
{
    Mat readImg, hsvImg;
    VideoCapture cap(1);
    if(!cap.isOpened())
    {
      cerr << "Cam cannot open" << endl;
      return -1;
    }

    int iLowH = 19;
    int iHighH = 27;

    int iLowS = 204;
    int iHighS = 255;

    int iLowV = 153;
    int iHighV = 255;

    namedWindow("Control");

    //Create trackbars in "Control" window
    createTrackbar("LowH", "Control", &iLowH, 179); //Hue (0 - 179)
    createTrackbar("HighH", "Control", &iHighH, 179);

    createTrackbar("LowS", "Control", &iLowS, 255); //Saturation (0 - 255)
    createTrackbar("HighS", "Control", &iHighS, 255);

    createTrackbar("LowV", "Control", &iLowV, 255);//Value (0 - 255)
    createTrackbar("HighV", "Control", &iHighV, 255);

#ifdef USE_GPU
    int gpuIdVal = gpu::getDevice();

    if(gpuIdVal >= 0)
    {
        gpu::printShortCudaDeviceInfo(gpuIdVal);
        gpu::setDevice(gpuIdVal);
    }
#endif

    while (1) {

      cv::TickMeter t_meter;
      t_meter.start();

      cap >> readImg;
      if(readImg.empty())
      {
        cerr << "Frame empty!" << endl;
        return -1;
      }

      Mat showImg = Mat::zeros(readImg.size(), readImg.type()), contoursImg = Mat::zeros(readImg.size(), readImg.type());

#ifdef USE_GPU
      gpu::GpuMat readGpuImg(readImg);
      gpu::GpuMat filteredImg, inRangeDst(readImg.rows, readImg.cols, CV_8UC1);

      if(readGpuImg.empty())
      {
          cerr << "Gpu Frame empty!" << endl;
          return -1;
      }

      gpu::cvtColor(readGpuImg, readGpuImg, COLOR_BGR2HSV);

      const int m = 16;
      int numRows = readGpuImg.rows, numCols = readGpuImg.cols;
      if (numRows == 0 || numCols == 0) return;

      const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
      const dim3 blockSize(m, m, 1);

      gpu_inRange<<<gridSize, blockSize>>>(readGpuImg, inRangeDst,
                                           iLowH, iHighH, iLowS, iHighS,
                                           iLowV, iHighV);
      //gpu_inRange<<<gridSize, blockSize>>>(readGpuImg, inRangeDst);


      hipDeviceSynchronize(); CudaCheckError();

      int k_size = 11;

      gpu::bilateralFilter(inRangeDst, filteredImg, k_size, 150, 150);

      filteredImg.download(showImg);
      readGpuImg.download(hsvImg);
#else
      Mat threshImg;
      cvtColor(readImg, hsvImg, CV_BGR2HSV);
      inRange(hsvImg, Scalar(iLowH, iLowS, iLowV), Scalar(iHighH, iHighS, iHighV), threshImg);

      erode(threshImg, threshImg, getStructuringElement(MORPH_ELLIPSE, Size(3, 3)));
      dilate(threshImg, threshImg, getStructuringElement(MORPH_ELLIPSE, Size(8, 8)));

      //morphological closing (fill small holes in the foreground)
      dilate(threshImg, threshImg, getStructuringElement(MORPH_ELLIPSE, Size(8, 8)));
      erode(threshImg, threshImg, getStructuringElement(MORPH_ELLIPSE, Size(3, 3)));

      showImg = threshImg.clone();
#endif

      trackObject(showImg, contoursImg, Scalar(0, 0, 255));

      //imshow("Org Frame", readImg);
      imshow("Obj Frame", readImg);
      //imshow("HSV Frame", hsvImg);
      imshow("Proc Frame", showImg);
      imshow("Contours Frame", contoursImg);

      t_meter.stop();

      cout << "FPS: " <<  1000 / (t_meter.getTimeMilli()) << endl;

      waitKey(10);
    }

    hipDeviceReset();

    return 0;
}
